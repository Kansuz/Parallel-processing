#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <list>
#include <cstdio>
#include <cmath>
#include <regex>
#include <chrono>

#define threads 4

//CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace std::chrono;

//TRANSKRYPCJA (DNA -> RNA)
__device__ void transcription(char* dna, char* rna, int index, int length_of_parts, int additional){  
  for(int i = 0; i < length_of_parts + additional; i++){
    //A-U
    if(dna[index * length_of_parts + i] == 'A'){
      rna[index * length_of_parts + i] = 'U';
    }
    //T-A
    else if(dna[index * length_of_parts + i] == 'T'){
      rna[index * length_of_parts + i] = 'A';
    } 
    //G-C
    else if(dna[index * length_of_parts + i] == 'G'){
      rna[index * length_of_parts + i] = 'C';
    }
    //C-G
    else if(dna[index * length_of_parts + i] == 'C'){
      rna[index * length_of_parts + i] = 'G';
    }
  }
}

__global__ void transform(char* dna, char* rna, char* result, int length_of_parts, int length, char* chain, char* code){
  int index = threadIdx.x;
  int additional = 0;

  if(index == threads - 1){
    additional = length - (length_of_parts * (threads - 1)) - length_of_parts;
  }
  transcription(dna, rna, index, length_of_parts, additional);
}

int main()
{
  string line, info;
  ifstream DNAFile("100.fasta");
  
  //ignorowanie pierwszej linijki, ponieważ są w niej zapisane dane genomu, a nie sam genom
  getline (DNAFile, info);

  //wczytywanie z pliku 
  string dna0, dna;
  while (getline (DNAFile, line)) {
    dna0 = dna0 + line;
  }
  for(char letter: dna0){
      dna.insert(0, string(1, letter));
  }

  char* charredDNA = (char*)malloc(sizeof(char) * dna.length());
  for(int i = 0; i < dna.length(); ++i){
    charredDNA[i] = dna[i];
  }
  charredDNA[dna.length()] = '\0';

  int length_of_parts = dna.length()/threads;

  char* cudaDNA;
  hipMalloc((void**)&cudaDNA, sizeof(char) * dna.length());
  char* cudaResult;
  hipMalloc((void**)&cudaResult, sizeof(char) * length_of_parts * threads);
  char* cudaRNA;
  hipMalloc((void**)&cudaRNA, sizeof(char) * length_of_parts * threads);
  char* cudaChain;
  hipMalloc((void**)&cudaChain, sizeof(char) * length_of_parts * threads / 3);
  char* cudaCode;
  hipMalloc((void**)&cudaCode, sizeof(char) * threads * 3);
  hipMemcpy(cudaDNA, charredDNA, sizeof(char) * dna.length(), hipMemcpyHostToDevice);

  auto start = high_resolution_clock::now();
  transform<<<1, threads>>>(cudaDNA, cudaRNA, cudaResult, length_of_parts, dna.length(), cudaChain, cudaCode);
  auto stop = high_resolution_clock::now();
  auto duration = duration_cast<microseconds>(stop - start);
  //cout << "Transcription took " << duration.count() << " microseconds." << '\n';

  hipFree(cudaDNA);
  free(charredDNA);

}
